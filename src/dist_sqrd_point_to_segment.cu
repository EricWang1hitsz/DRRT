#include "hip/hip_runtime.h"
#include "DRRT/dist_sqrd_point_to_segment.cuh"

__global__ void CUDADistanceSquaredPointToSegment(double* query_point_x,
                                                  double* query_point_y,
                                                  double* seg_start_x,
                                                  double* seg_start_y,
                                                  double* seg_end_x,
                                                  double* seg_end_y,
                                                  double* dist)
{
    int idx = threadIdx.x;
    double qpx = *query_point_x;
    double qpy = *query_point_y;
    double spx = seg_start_x[idx];
    double spy = seg_start_y[idx];
    double epx = seg_end_x[idx];
    double epy = seg_end_y[idx];
    double vx = qpx - spx;
    double vy = qpy - spy;
    double ux = epx - spx;
    double uy = epy - spy;

    double determinate = vx*ux + vy*uy;

    if( determinate <= 0 ) {
        dist[idx] = vx*vx + vy*vy;
    } else {
        double len = ux*ux + uy*uy;
        if( determinate >= len ) {
            dist[idx] = (epx-qpx)*(epx-qpx) + (epy-qpy)*(epy-qpy);
        } else {
            dist[idx] = (ux*vy - uy*vx)*(ux*vy - uy*vx) / len;
        }
    }
}

std::vector<double> CalcDistanceSquaredPointToSegment(
        std::vector<double> query_point,
        std::vector<std::vector<double>> starts,
        std::vector<std::vector<double>> ends)
{
    // Number of threads to use
    int grid_dim = 1;
    int block_dim = starts.size();

    // Host memory pointers
    double* h_qp_x = NULL;
    *h_qp_x = query_point[0];
    double* h_qp_y = NULL;
    *h_qp_y = query_point[1];
    double* h_start_x = NULL;
    double* h_start_y = NULL;
    double* h_end_x = NULL;
    double* h_end_y = NULL;
    for(int i = 0; i < (int)starts.size(); i++) {
        h_start_x[i] = starts.at(i).at(0);
        h_start_y[i] = starts.at(i).at(1);
        h_end_x[i] = ends.at(i).at(0);
        h_end_y[i] = ends.at(i).at(1);
    }
    double* h_dist = NULL;

    // Device memory pointers
    double* d_qp_x;
    double* d_qp_y;
    double* d_start_x;
    double* d_start_y;
    double* d_end_x;
    double* d_end_y;
    double* d_dist;

    // Allocate Device memory
    hipMalloc((void**) &d_qp_x, sizeof(double));
    hipMalloc((void**) &d_qp_y, sizeof(double));
    hipMalloc((void**) &d_start_x, sizeof(double) * starts.size());
    hipMalloc((void**) &d_start_y, sizeof(double) * starts.size());
    hipMalloc((void**) &d_end_x, sizeof(double) * ends.size());
    hipMalloc((void**) &d_end_y, sizeof(double) * ends.size());
    hipMalloc((void**) &d_dist, sizeof(double) * block_dim);

    // Transfer structures to Device
    hipMemcpy(d_qp_x, h_qp_x, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_qp_y, h_qp_y, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_start_x, h_start_x, sizeof(double) * starts.size(),
                               hipMemcpyHostToDevice);
    hipMemcpy(d_start_y, h_start_y, sizeof(double) * starts.size(),
                               hipMemcpyHostToDevice);
    hipMemcpy(d_end_x, h_end_x, sizeof(double) * ends.size(),
                               hipMemcpyHostToDevice);
    hipMemcpy(d_end_y, h_end_y, sizeof(double) * ends.size(),
                               hipMemcpyHostToDevice);

    // Launch kernel
    CUDADistanceSquaredPointToSegment<<<grid_dim,block_dim>>>(d_qp_x,
                                                                  d_qp_y,
                                                                  d_start_x,
                                                                  d_start_y,
                                                                  d_end_x,
                                                                  d_end_y,
                                                                  d_dist);

    // Transfer result back to Host
    hipMemcpy(h_dist, d_dist, sizeof(double) * block_dim,
                               hipMemcpyDeviceToHost);

    // Put results into vector and return for use
    std::vector<double> dists(block_dim);
    for(int i = 0; i < block_dim; i++) {
        dists.at(i) = ((double)h_dist[i]);
    }
    return dists;
}
