#include "hip/hip_runtime.h"
#include "DRRT/dist_sqrd_point_to_segment.cuh"

__global__ void CUDADistanceSquaredPointToSegment(double* query_point_x,
                                                  double* query_point_y,
                                                  double* seg_start_x,
                                                  double* seg_start_y,
                                                  double* seg_end_x,
                                                  double* seg_end_y,
                                                  double* dist)
{
    int idx = threadIdx.x;
    double qpx = *query_point_x;
    double qpy = *query_point_y;
    double spx = seg_start_x[idx];
    double spy = seg_start_y[idx];
    double epx = seg_end_x[idx];
    double epy = seg_end_y[idx];
    double vx = qpx - spx;
    double vy = qpy - spy;
    double ux = epx - spx;
    double uy = epy - spy;

    double determinate = vx*ux + vy*uy;

    if( determinate <= 0 ) {
        dist[idx] = vx*vx + vy*vy;
    } else {
        double len = ux*ux + uy*uy;
        if( determinate >= len ) {
            dist[idx] = (epx-qpx)*(epx-qpx) + (epy-qpy)*(epy-qpy);
        } else {
            dist[idx] = (ux*vy - uy*vx)*(ux*vy - uy*vx) / len;
        }
    }
}

std::vector<double> CalcDistanceSquaredPointToSegment(
        std::vector<double> query_point,
        std::vector<std::vector<double>> starts,
        std::vector<std::vector<double>> ends)
{
    // Number of threads to use
    int grid_dim = 1;
    int block_dim = starts[0].size();

    // Host memory pointers
//    std::cout << "host pointers" << std::endl;
    double* h_qp_x = nullptr;
    h_qp_x = &(query_point[0]);
    double* h_qp_y = nullptr;
    h_qp_y = &(query_point[1]);
    std::vector<double> h_start_x;
    std::vector<double> h_start_y;
    std::vector<double> h_end_x;
    std::vector<double> h_end_y;
    for(int i = 0; i < block_dim; i++) {
        h_start_x.push_back(starts[i][0]);
        h_start_y.push_back(starts[i][1]);
        h_end_x.push_back(ends[i][0]);
        h_end_y.push_back(ends[i][1]);
    }
    double h_dist[block_dim];

    // Device memory pointers
//    std::cout << "device pointers" << std::endl;
    double* d_qp_x = nullptr;
    double* d_qp_y = nullptr;
    double* d_start_x = nullptr;
    double* d_start_y = nullptr;
    double* d_end_x = nullptr;
    double* d_end_y = nullptr;
    double* d_dist = nullptr;

    // Allocate Device memory
//    std::cout << "hipMalloc" << std::endl;
    hipMalloc((void**) &d_qp_x, sizeof(double));
    hipMalloc((void**) &d_qp_y, sizeof(double));
    hipMalloc((void**) &d_start_x, sizeof(double) * h_start_x.size());
    hipMalloc((void**) &d_start_y, sizeof(double) * h_start_y.size());
    hipMalloc((void**) &d_end_x, sizeof(double) * h_end_x.size());
    hipMalloc((void**) &d_end_y, sizeof(double) * h_end_y.size());
    hipMalloc((void**) &d_dist, sizeof(double) * block_dim);

    // Transfer structures to Device
//    std::cout << "hipMemcpyHostToDevice" << std::endl;
    hipMemcpy(d_qp_x, h_qp_x, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_qp_y, h_qp_y, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_start_x, &h_start_x[0], sizeof(double) * h_start_x.size(),
                               hipMemcpyHostToDevice);
    hipMemcpy(d_start_y, &h_start_y[0], sizeof(double) * h_start_y.size(),
                               hipMemcpyHostToDevice);
    hipMemcpy(d_end_x, &h_end_x[0], sizeof(double) * h_end_x.size(),
                               hipMemcpyHostToDevice);
    hipMemcpy(d_end_y, &h_end_y[0], sizeof(double) * h_end_y.size(),
                               hipMemcpyHostToDevice);

    // Launch kernel
//    std::cout << "kernel" << std::endl;
    CUDADistanceSquaredPointToSegment<<<grid_dim,block_dim>>>(d_qp_x,
                                                              d_qp_y,
                                                              d_start_x,
                                                              d_start_y,
                                                              d_end_x,
                                                              d_end_y,
                                                              d_dist);

    // Transfer result back to Host
//    std::cout << "hipMemcpyDeviceToHost" << std::endl;
    hipMemcpy(&h_dist[0], d_dist, sizeof(double) * block_dim,
                               hipMemcpyDeviceToHost);

    // Put results into vector and return for use
    std::vector<double> dists(block_dim);
    for(int i = 0; i < block_dim; i++) {
        dists.at(i) = ((double)h_dist[i]);
    }
    return dists;
}
