#include "hip/hip_runtime.h"
#include <stdio.h>

// query_point_x holds all query_point x-values
__global__ double DistanceSqrdPointToSegment(double* query_point_x,
                                             double* query_point_y,
                                             double* seg_start_x,
                                             double* seg_start_y,
                                             double* seg_end_x,
                                             double* seg_end_y)
{
    int idx = 1;//threadIdx.x;
    double qpx = query_point_x[idx];
    double qpy = query_point_y[idx];
    double spx = seg_start_x[idx];
    double spy = seg_start_y[idx];
    double epx = seg_end_x[idx];
    double epy = seg_end_y[idx];
    double vx = qpx - spx;
    double vy = qpy - spy;
    double ux = epx - spx;
    double uy = epy - spy;

    double determinate = vx*ux + vy*uy;

    if( determinate <= 0 ) {
        return vx*vx + vy*vy;
    } else {
        double len = ux*ux + uy*uy;
        if( determinate >= len ) {
            return (epx-qpx)*(epx-qpx) + (epy-qpy)*(epy-qpy);
        } else {
            return (ux*vy - uy*vx)*(ux*vy - uy*vx) / len;
        }
    }
}
